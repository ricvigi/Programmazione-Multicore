#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
    printf("Hello, World\n");
}

int main(int argc, char** argv)
{
    hello<<<1,10>>>();
    hipDeviceSynchronize();
    return 0;
}
