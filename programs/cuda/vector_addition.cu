#include <stdio.h>
#include <hip/hip_runtime.h>
#define MAX 100
#define CUDA_CHECK_RETURN(value) do { \
    hipError_t _m_cudaStat = value; \
    if (_m_cudaStat != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n",\
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1); \
    } \
} while(0)

void vecAdd (float* A, float* B, float* C, int n);
__global__ void vecAddKernel(float* A, float* B, float* C, int n);
float* create_random_vector(int n, int m);

int main(int argc, char** argv)
{
    int n = atoi(argv[1]);
    float* A = create_random_vector(n, MAX);
    float* B = create_random_vector(n, MAX);
    float* C = create_random_vector(n, MAX);

    vecAdd(A, B, C, n);
    for (int i = 0; i < n; i++) {
        printf("A[%d]%f + B[%d]%f = C[%d]%f", i, A[i], i, B[i], i, C[i]);
    }

    return EXIT_SUCCESS;
}

__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

void vecAdd (float* A, float* B, float* C, int n)
{
    int size = n * sizeof(float);
    float* d_A, *d_B, *d_C;

    CUDA_CHECK_RETURN(hipMalloc((void**) &d_A, size));
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    CUDA_CHECK_RETURN(hipMalloc((void**) &d_B, size));
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    CUDA_CHECK_RETURN(hipMalloc((void**) &d_C, size));

    vecAddKernel<<<((n + 255) / 256),256>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    CUDA_CHECK_RETURN(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

float* create_random_vector(int n, int m) {
    float* vec = (float*) malloc(n * sizeof(float));
    for(int i = 0; i < n; i++)
    {
        vec[i] = (float) (rand() % m);
    }
    return vec;
}
