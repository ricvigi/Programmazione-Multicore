#include <stdio.h>
#include <hip/hip_runtime.h>

void countDevices()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) printf("No CUDA compatible GPU esists.\n");
    else
    {
        hipDeviceProp_t pr;
        for (int i = 0 ; i < deviceCount; i++)
        {
            hipGetDeviceProperties(&pr, i);
            printf("Dev #%i is %s\n", i, pr.name);
        }
    }
}

int main(int argc, char** argv) {
    countDevices();
    return 0;
}
